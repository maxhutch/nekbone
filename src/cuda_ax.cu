#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "magma.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define MIN(a,b) (((a)<(b))?(a):(b))
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
       fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
       if (abort) exit(code);
   }
}


static __global__ void transform_k(double* ur,
                                   double* us,
				   double* ut,
				   double* trans,
				   int n){

  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  const int nthreads = blockDim.x * gridDim.x;
  double wr, ws, wt;
  int i;

  for (i = idx; i < n; i += nthreads){
    wr = trans[6*i+0]*ur[i] + trans[6*i+1] * us[i] + trans[6*i+2]*ut[i];
    ws = trans[6*i+1]*ur[i] + trans[6*i+3] * us[i] + trans[6*i+4]*ut[i];
    wt = trans[6*i+2]*ur[i] + trans[6*i+4] * us[i] + trans[6*i+5]*ut[i];
    ur[i] = wr;
    us[i] = ws;
    ut[i] = wt;
  }
}

static __global__ void set_addr(double** batch_u,
                                double** batch_us,
				double** batch_u2,
				double** batch_ut,
				double** batch_Dt,
				double** batch_D,
				double* u,
				double* us,
				double* ut,
				double* Dt,
				double* D,
				int size,
				int num){
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  const int nthreads = blockDim.x * gridDim.x;
  int i;
  for (i = idx; i < num; i+=nthreads){
    batch_u[i]  = u  + i*size*size;
    batch_us[i] = us + i*size*size;
    batch_Dt[i] = Dt;
    batch_D[i]  = D;
    if (i*size < num) {
    batch_u2[i] = u  + i*size*size*size;
    batch_ut[i] = ut + i*size*size*size;
    }
  }
}

#define NUM_BLOCK_MAX 16
#define NUM_STREAM_MAX 8
#define USE_BATCH
extern hipStream_t* streams;
extern hipblasHandle_t cublas_ctx;
static double* g_d;
extern "C" void ax_e_cuda_(double *w,
                                      double* u,
                                      double* D,
				      double* Dt,
				      double* g,
				      int* n,
				      int* m){
  int size1 = *n + 1;
  int size2 = size1 * size1;
  int size3 = size1 * size1 * size1;
  int num = *m;
  int i, j, jp;

  int num_block_l;
 
  const double zero = 0.0, one = 1.0;

  double *u_d, *D_d, *Dt_d;
  double *ur_d, *us_d, *ut_d; 
  double *u_l, *ur_l, *us_l, *ut_l, *g_l;
  double *u_h, *w_h;

  double **batch_Dt_d; double **batch_D_d;
  double **batch_u_d;  double **batch_u_l; 
  double **batch_u2_d; double **batch_u2_l; 
  double **batch_us_d; double **batch_us_l; 
  double **batch_ut_d; double **batch_ut_l; 


  // First, copy over D, Dt
  gpuErrchk(hipMalloc(&D_d , size2 * sizeof(double)))
  gpuErrchk(hipMalloc(&Dt_d, size2 * sizeof(double)))
  gpuErrchk(hipMemcpy(D_d , D , size2*sizeof(double), hipMemcpyHostToDevice))
#if 1
  gpuErrchk(hipMemcpy(Dt_d, Dt, size2*sizeof(double), hipMemcpyHostToDevice))
#else
  gpuErrchk(hipMemcpy(Dt_d, D_d, size2*sizeof(double), hipMemcpyDeviceToDevice)) 
  magmablas_dtranspose_inplace(size1, Dt_d, size1);
#endif

// Allocate space for other stuff
  gpuErrchk(hipMalloc(&ur_d, NUM_BLOCK_MAX * NUM_STREAM_MAX * size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&us_d, NUM_BLOCK_MAX * NUM_STREAM_MAX * size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&ut_d, NUM_BLOCK_MAX * NUM_STREAM_MAX * size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&u_d , NUM_BLOCK_MAX * NUM_STREAM_MAX * size3 * sizeof(double)))

  gpuErrchk(hipMalloc(&batch_u_d , NUM_BLOCK_MAX * NUM_STREAM_MAX * size1 * sizeof(double*)))
  gpuErrchk(hipMalloc(&batch_us_d , NUM_BLOCK_MAX * NUM_STREAM_MAX * size1 * sizeof(double*)))
  gpuErrchk(hipMalloc(&batch_u2_d , NUM_BLOCK_MAX * NUM_STREAM_MAX * sizeof(double*)))
  gpuErrchk(hipMalloc(&batch_ut_d , NUM_BLOCK_MAX * NUM_STREAM_MAX * sizeof(double*)))
  gpuErrchk(hipMalloc(&batch_Dt_d , NUM_BLOCK_MAX * size1 * sizeof(double*)))
  gpuErrchk(hipMalloc(&batch_D_d , NUM_BLOCK_MAX * size1 * sizeof(double*)))

  u_h = u; w_h = w;
  // Loop over blocks
  for (j = 0; j < num; j+=NUM_BLOCK_MAX){ 
    jp = (j/NUM_BLOCK_MAX) % NUM_STREAM_MAX; // stream ID
    num_block_l = MIN(NUM_BLOCK_MAX, num-j); // number of elements in block
    u_l  =  u_d + jp*NUM_BLOCK_MAX*size3; ur_l = ur_d + jp*NUM_BLOCK_MAX*size3; 
    us_l = us_d + jp*NUM_BLOCK_MAX*size3; ut_l = ut_d + jp*NUM_BLOCK_MAX*size3;
    g_l  =  g_d + j*6*size3;
    batch_u_l  = batch_u_d  + jp*NUM_BLOCK_MAX*size1; batch_us_l = batch_us_d + jp*NUM_BLOCK_MAX*size1; 
    batch_u2_l = batch_u2_d + jp*NUM_BLOCK_MAX; batch_ut_l = batch_ut_d + jp*NUM_BLOCK_MAX; 

    // copy over u
    gpuErrchk(hipMemcpyAsync(u_l, u_h+  j*size3,   size3*sizeof(double)*num_block_l, hipMemcpyHostToDevice, streams[jp]))
#ifdef USE_BATCH
    set_addr<<<1,32, 0, streams[jp]>>>(batch_u_l, batch_us_l, batch_u2_l, batch_ut_l,
                                       batch_Dt_d, batch_D_d,
                                       u_l, us_l, ut_l, Dt_d, D_d, 
				       size1, size1*num_block_l);
#endif
    // dgemm('N','N',n1,n3,n2,1.0,a,n1,b,n2,0.0,c,n1)
    magmablasSetKernelStream(streams[jp]);
    hipblasSetStream(cublas_ctx, streams[jp]);
    magma_dgemm('T', 'N', size1, size2*num_block_l, size1, 
                1.0,  Dt_d, size1, 
                      u_l, size1,
                0.0, ur_l, size1);
#ifdef USE_BATCH
    hipblasDgemmBatched(cublas_ctx,
                       HIPBLAS_OP_N, HIPBLAS_OP_N, size1, size1, size1,
		       &one, (const double **) batch_u_l, size1,
		             (const double **) batch_Dt_d, size1,
		       &zero, batch_us_l, size1,
		           size1*num_block_l);
    hipblasDgemmBatched(cublas_ctx,
                       HIPBLAS_OP_N, HIPBLAS_OP_N, size2, size1, size1,
		       &one, (const double **) batch_u2_l, size2,
		             (const double **) batch_Dt_d, size1,
		       &zero, batch_ut_l, size2,
		             num_block_l);
#else
    for (i = 0; i < size1*num_block_l; i++){
      magma_dgemm('N', 'N', size1, size1, size1,
                  1.0,  u_l + i*size2, size1,
                       Dt_d          , size1,
                  0.0, us_l + i*size2, size1);
    }
    for (i = 0;i < num_block_l; i++){
    magma_dgemm('N', 'N', size2, size1, size1,
                1.0, u_l +i*size3, size2,
                     Dt_d, size1,
	        0.0, ut_l +i*size3, size2);
    }
#endif
    transform_k<<<128, 256, 0, streams[jp]>>>(ur_l, us_l, ut_l, g_l, size3*num_block_l);

    magma_dgemm('T', 'N', size1, size2*num_block_l, size1, 
                1.0, D_d, size1, 
                     ur_l, size1,
                0.0,  u_l, size1);
#ifdef USE_BATCH
    hipblasDgemmBatched(cublas_ctx,
                       HIPBLAS_OP_N, HIPBLAS_OP_N, size1, size1, size1,
		       &one, (const double **) batch_us_l, size1,
		             (const double **) batch_D_d, size1,
		       &one, batch_u_l, size1,
		             size1*num_block_l);
    hipblasDgemmBatched(cublas_ctx,
                       HIPBLAS_OP_N, HIPBLAS_OP_N, size2, size1, size1,
		       &one, (const double **) batch_ut_l, size2,
		             (const double **) batch_D_d, size1,
		       &one, batch_u2_l, size2,
		             num_block_l);
#else
    for (i = 0; i < size1*num_block_l; i++){
      magma_dgemm('N', 'N', size1, size1, size1,
                  1.0, us_l + i*size2, size1,
                        D_d          , size1,
	          1.0,  u_l + i*size2, size1);
    }
    for (i = 0; i < num_block_l; i++){
    magma_dgemm('N', 'N', size2, size1, size1,
                1.0, ut_l + i*size3, size2,
                      D_d, size1,
	        1.0,  u_l + i*size3, size2);
    }
#endif
   // Copy back w
    gpuErrchk(hipMemcpyAsync(w_h+j*size3, u_l, size3*sizeof(double)*num_block_l, hipMemcpyDeviceToHost, streams[jp]))
  }
  hipDeviceSynchronize(); 

  gpuErrchk(hipFree(ur_d));
  gpuErrchk(hipFree(us_d));
  gpuErrchk(hipFree(ut_d));
  gpuErrchk(hipFree( u_d));
  gpuErrchk(hipFree( D_d));
  gpuErrchk(hipFree(Dt_d));
  gpuErrchk(hipFree(batch_u_d));
  gpuErrchk(hipFree(batch_u2_d));
  gpuErrchk(hipFree(batch_us_d));
  gpuErrchk(hipFree(batch_ut_d));
  gpuErrchk(hipFree(batch_Dt_d));
  gpuErrchk(hipFree(batch_D_d))
}

extern "C" void setup_cg_cuda_(double* w,
                          double* u,
			  double* g,
			  int* n,
			  int* m){

  int size1 = *n + 1;
  int size3 = size1 * size1 * size1;
  int num = *m; 
  
  // Pin memcpy buffers
  gpuErrchk(hipHostRegister(u, num*size3*sizeof(double), 0));
  gpuErrchk(hipHostRegister(w, num*size3*sizeof(double), 0));
  gpuErrchk(hipHostRegister(g, num*6*size3*sizeof(double), 0));
 
  // Copy over g
  gpuErrchk(hipMalloc(&g_d,    num * 6 * size3 * sizeof(double)))
  gpuErrchk(hipMemcpy(g_d , g, num *6*size3*sizeof(double), hipMemcpyHostToDevice))

}

extern "C" void teardown_cg_cuda_(double* w,
                             double* u,
			     double* g,
			     int* n,
			     int* m){

  // Un-pin the buffers (so it can be paged again)
  gpuErrchk(hipHostUnregister(u));
  gpuErrchk(hipHostUnregister(w));
  gpuErrchk(hipHostUnregister(g));

  // Free the device copy of g
  gpuErrchk(hipFree(g_d));
}

