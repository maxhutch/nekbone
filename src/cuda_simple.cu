#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "magma.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
       fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
       if (abort) exit(code);
   }
}

extern "C" void local_grad3_cuda_(double *ur,
                                 double *us, 
				 double *ut,
				 double *u,
				 int *n,
				 double *D,
				 double *Dt){

  int size1 = *n + 1;
  int size2 = size1 * size1;
  int size3 = size1 * size1 * size1;
  int i;

  double *ur_d, *us_d, *ut_d, *u_d, *D_d, *Dt_d;
  gpuErrchk(hipMalloc(&ur_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&us_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&ut_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&u_d , size3 * sizeof(double)))

  gpuErrchk(hipMalloc(&D_d , size2 * sizeof(double)))
  gpuErrchk(hipMalloc(&Dt_d, size2 * sizeof(double)))

  gpuErrchk(hipMemcpy(u_d , u , size3*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(D_d , D , size2*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(Dt_d, Dt, size2*sizeof(double), hipMemcpyHostToDevice))

  // dgemm('N','N',n1,n3,n2,1.0,a,n1,b,n2,0.0,c,n1)
  magma_dgemm('N', 'N', size1, size2, size1, 
              1.0,  D_d, size1, 
	            u_d, size1,
              0.0, ur_d, size1);
  for (i = 0; i < size1; i++){
    magma_dgemm('N', 'N', size1, size1, size1,
                1.0,  u_d + i*size2, size1,
		     Dt_d          , size1,
		0.0, us_d + i*size2, size1);
  }
  magma_dgemm('N', 'N', size2, size1, size1,
              1.0, u_d , size2,
	           Dt_d, size1,
	      0.0, ut_d, size2);

  gpuErrchk(hipMemcpy(ur, ur_d, size3*sizeof(double), hipMemcpyDeviceToHost))
  gpuErrchk(hipMemcpy(us, us_d, size3*sizeof(double), hipMemcpyDeviceToHost))
  gpuErrchk(hipMemcpy(ut, ut_d, size3*sizeof(double), hipMemcpyDeviceToHost))
  
  gpuErrchk(hipFree(ur_d)) gpuErrchk(hipFree(us_d)) gpuErrchk(hipFree(ut_d)) gpuErrchk(hipFree(u_d))
  gpuErrchk(hipFree(D_d)) gpuErrchk(hipFree(Dt_d))
}

extern "C" void local_grad3_t_cuda_(double *u,
                                    double *ur,
                                    double *us, 
				    double *ut,
				    int *n,
				    double *D,
				    double *Dt,
				    double *w){

  int size1 = *n + 1;
  int size2 = size1 * size1;
  int size3 = size1 * size1 * size1;
  int i;

  double *ur_d, *us_d, *ut_d, *u_d, *D_d, *Dt_d;
  gpuErrchk(hipMalloc(&ur_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&us_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&ut_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&u_d , size3 * sizeof(double)))

  gpuErrchk(hipMalloc(&D_d , size2 * sizeof(double)))
  gpuErrchk(hipMalloc(&Dt_d, size2 * sizeof(double)))

  gpuErrchk(hipMemcpy(ur_d, ur, size3*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(us_d, us, size3*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(ut_d, ut, size3*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(D_d , D , size2*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(Dt_d, Dt, size2*sizeof(double), hipMemcpyHostToDevice))

  // dgemm('N','N',n1,n3,n2,1.0,a,n1,b,n2,0.0,c,n1)
  magma_dgemm('N', 'N', size1, size2, size1, 
              1.0, Dt_d, size1, 
	           ur_d, size1,
              0.0,  u_d, size1);
  for (i = 0; i < size1; i++){
    magma_dgemm('N', 'N', size1, size1, size1,
                1.0, us_d + i*size2, size1,
		      D_d          , size1,
		1.0,  u_d + i*size2, size1);
  }
  magma_dgemm('N', 'N', size2, size1, size1,
              1.0, ut_d, size2,
	            D_d, size1,
	      1.0,  u_d, size2);

  gpuErrchk(hipMemcpy(u, u_d, size3*sizeof(double), hipMemcpyDeviceToHost))
  
  gpuErrchk(hipFree(ur_d)) gpuErrchk(hipFree(us_d)) gpuErrchk(hipFree(ut_d)) gpuErrchk(hipFree(u_d))
  gpuErrchk(hipFree(D_d)) gpuErrchk(hipFree(Dt_d))
}

static __global__ void transform_k(double* ur,
                                   double* us,
				   double* ut,
				   double* trans,
				   int n){

  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  const int nthreads = blockDim.x * gridDim.x;
  double wr, ws, wt;
  int i;

  for (i = idx; i < n; i += nthreads){
    wr = trans[6*i+0]*ur[i] + trans[6*i+1] * us[i] + trans[6*i+2]*ut[i];
    ws = trans[6*i+1]*ur[i] + trans[6*i+3] * us[i] + trans[6*i+4]*ut[i];
    wt = trans[6*i+2]*ur[i] + trans[6*i+4] * us[i] + trans[6*i+5]*ut[i];
    ur[i] = wr;
    us[i] = ws;
    ut[i] = wt;
  }
}


extern "C" void local_grad3_comb_cuda_(double *w,
                                      double* u,
                                      double* D,
				      double* Dt,
				      double* g,
				      int* n){
  int size1 = *n + 1;
  int size2 = size1 * size1;
  int size3 = size1 * size1 * size1;
  int i;

  double *u_d, *D_d, *Dt_d, *g_d;
  double *ur_d, *us_d, *ut_d; 

  gpuErrchk(hipMalloc(&ur_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&us_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&ut_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&u_d , size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&g_d , 6 * size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&D_d , size2 * sizeof(double)))
  gpuErrchk(hipMalloc(&Dt_d, size2 * sizeof(double)))

  gpuErrchk(hipMemcpy(u_d , u , size3*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(D_d , D , size2*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(Dt_d, Dt, size2*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(g_d , g , 6*size3*sizeof(double), hipMemcpyHostToDevice))

  // dgemm('N','N',n1,n3,n2,1.0,a,n1,b,n2,0.0,c,n1)
  magma_dgemm('N', 'N', size1, size2, size1, 
              1.0,  D_d, size1, 
	            u_d, size1,
              0.0, ur_d, size1);
  for (i = 0; i < size1; i++){
    magma_dgemm('N', 'N', size1, size1, size1,
                1.0,  u_d + i*size2, size1,
		     Dt_d          , size1,
		0.0, us_d + i*size2, size1);
  }
  magma_dgemm('N', 'N', size2, size1, size1,
              1.0, u_d , size2,
	           Dt_d, size1,
	      0.0, ut_d, size2);

  transform_k<<<128, 256>>>(ur_d, us_d, ut_d, g_d, size3);

  magma_dgemm('N', 'N', size1, size2, size1, 
              1.0, Dt_d, size1, 
	           ur_d, size1,
              0.0,  u_d, size1);
  for (i = 0; i < size1; i++){
    magma_dgemm('N', 'N', size1, size1, size1,
                1.0, us_d + i*size2, size1,
		      D_d          , size1,
		1.0,  u_d + i*size2, size1);
  }
  magma_dgemm('N', 'N', size2, size1, size1,
              1.0, ut_d, size2,
	            D_d, size1,
	      1.0,  u_d, size2);

  gpuErrchk(hipMemcpy(w, u_d, size3*sizeof(double), hipMemcpyDeviceToHost))
  
  gpuErrchk(hipFree(ur_d)) gpuErrchk(hipFree(us_d)) gpuErrchk(hipFree(ut_d)) gpuErrchk(hipFree(u_d))
  gpuErrchk(hipFree(D_d)) gpuErrchk(hipFree(Dt_d)) gpuErrchk(hipFree(g_d))
}


extern "C" void ax_e_cuda_(double *w,
                                      double* u,
                                      double* D,
				      double* Dt,
				      double* g,
				      int* n,
				      int* m){
  int size1 = *n + 1;
  int size2 = size1 * size1;
  int size3 = size1 * size1 * size1;
  int num = *m;
  int i, j;

  double *u_d, *D_d, *Dt_d, *g_d;
  double *ur_d, *us_d, *ut_d; 

  // First, copy over D, Dt
  gpuErrchk(hipMalloc(&D_d , size2 * sizeof(double)))
  gpuErrchk(hipMalloc(&Dt_d, size2 * sizeof(double)))
  gpuErrchk(hipMemcpy(D_d , D , size2*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(Dt_d, Dt, size2*sizeof(double), hipMemcpyHostToDevice))

  // Allocate space for other stuff
  gpuErrchk(hipMalloc(&ur_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&us_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&ut_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&u_d , size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&g_d , 6 * size3 * sizeof(double)))

  for (j = 0; j < num; j++){ 
    gpuErrchk(hipMemcpy(u_d , u+  j*size3,   size3*sizeof(double), hipMemcpyHostToDevice))
    gpuErrchk(hipMemcpy(g_d , g+6*j*size3, 6*size3*sizeof(double), hipMemcpyHostToDevice))

    // dgemm('N','N',n1,n3,n2,1.0,a,n1,b,n2,0.0,c,n1)
    magma_dgemm('N', 'N', size1, size2, size1, 
                1.0,  D_d, size1, 
                      u_d, size1,
                0.0, ur_d, size1);
    for (i = 0; i < size1; i++){
      magma_dgemm('N', 'N', size1, size1, size1,
                  1.0,  u_d + i*size2, size1,
                       Dt_d          , size1,
                  0.0, us_d + i*size2, size1);
    }
    magma_dgemm('N', 'N', size2, size1, size1,
                1.0, u_d , size2,
                     Dt_d, size1,
	        0.0, ut_d, size2);

    transform_k<<<128, 256>>>(ur_d, us_d, ut_d, g_d, size3);

    magma_dgemm('N', 'N', size1, size2, size1, 
                1.0, Dt_d, size1, 
                     ur_d, size1,
                0.0,  u_d, size1);
    for (i = 0; i < size1; i++){
      magma_dgemm('N', 'N', size1, size1, size1,
                  1.0, us_d + i*size2, size1,
                        D_d          , size1,
	          1.0,  u_d + i*size2, size1);
    }
    magma_dgemm('N', 'N', size2, size1, size1,
                1.0, ut_d, size2,
                      D_d, size1,
	        1.0,  u_d, size2);
    gpuErrchk(hipMemcpy(w+j*size3, u_d, size3*sizeof(double), hipMemcpyDeviceToHost))
  }
  
  gpuErrchk(hipFree(ur_d)) gpuErrchk(hipFree(us_d)) gpuErrchk(hipFree(ut_d)) gpuErrchk(hipFree(u_d))
  gpuErrchk(hipFree(D_d)) gpuErrchk(hipFree(Dt_d)) gpuErrchk(hipFree(g_d))
}





