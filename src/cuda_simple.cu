#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "magma.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define MIN(a,b) (((a)<(b))?(a):(b))
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
       fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
       if (abort) exit(code);
   }
}


extern "C" void local_grad3_cuda_(double *ur,
                                 double *us, 
				 double *ut,
				 double *u,
				 int *n,
				 double *D,
				 double *Dt){

  int size1 = *n + 1;
  int size2 = size1 * size1;
  int size3 = size1 * size1 * size1;
  int i;

  double *ur_d, *us_d, *ut_d, *u_d, *D_d, *Dt_d;
  gpuErrchk(hipMalloc(&ur_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&us_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&ut_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&u_d , size3 * sizeof(double)))

  gpuErrchk(hipMalloc(&D_d , size2 * sizeof(double)))
  gpuErrchk(hipMalloc(&Dt_d, size2 * sizeof(double)))

  gpuErrchk(hipMemcpy(u_d , u , size3*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(D_d , D , size2*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(Dt_d, Dt, size2*sizeof(double), hipMemcpyHostToDevice))

  // dgemm('N','N',n1,n3,n2,1.0,a,n1,b,n2,0.0,c,n1)
  magma_dgemm('N', 'N', size1, size2, size1, 
              1.0,  D_d, size1, 
	            u_d, size1,
              0.0, ur_d, size1);
  for (i = 0; i < size1; i++){
    magma_dgemm('N', 'N', size1, size1, size1,
                1.0,  u_d + i*size2, size1,
		     Dt_d          , size1,
		0.0, us_d + i*size2, size1);
  }
  magma_dgemm('N', 'N', size2, size1, size1,
              1.0, u_d , size2,
	           Dt_d, size1,
	      0.0, ut_d, size2);

  gpuErrchk(hipMemcpy(ur, ur_d, size3*sizeof(double), hipMemcpyDeviceToHost))
  gpuErrchk(hipMemcpy(us, us_d, size3*sizeof(double), hipMemcpyDeviceToHost))
  gpuErrchk(hipMemcpy(ut, ut_d, size3*sizeof(double), hipMemcpyDeviceToHost))
  
  gpuErrchk(hipFree(ur_d)) gpuErrchk(hipFree(us_d)) gpuErrchk(hipFree(ut_d)) gpuErrchk(hipFree(u_d))
  gpuErrchk(hipFree(D_d)) gpuErrchk(hipFree(Dt_d))
}

extern "C" void local_grad3_t_cuda_(double *u,
                                    double *ur,
                                    double *us, 
				    double *ut,
				    int *n,
				    double *D,
				    double *Dt,
				    double *w){

  int size1 = *n + 1;
  int size2 = size1 * size1;
  int size3 = size1 * size1 * size1;
  int i;

  double *ur_d, *us_d, *ut_d, *u_d, *D_d, *Dt_d;
  gpuErrchk(hipMalloc(&ur_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&us_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&ut_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&u_d , size3 * sizeof(double)))

  gpuErrchk(hipMalloc(&D_d , size2 * sizeof(double)))
  gpuErrchk(hipMalloc(&Dt_d, size2 * sizeof(double)))

  gpuErrchk(hipMemcpy(ur_d, ur, size3*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(us_d, us, size3*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(ut_d, ut, size3*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(D_d , D , size2*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(Dt_d, Dt, size2*sizeof(double), hipMemcpyHostToDevice))

  // dgemm('N','N',n1,n3,n2,1.0,a,n1,b,n2,0.0,c,n1)
  magma_dgemm('N', 'N', size1, size2, size1, 
              1.0, Dt_d, size1, 
	           ur_d, size1,
              0.0,  u_d, size1);
  for (i = 0; i < size1; i++){
    magma_dgemm('N', 'N', size1, size1, size1,
                1.0, us_d + i*size2, size1,
		      D_d          , size1,
		1.0,  u_d + i*size2, size1);
  }
  magma_dgemm('N', 'N', size2, size1, size1,
              1.0, ut_d, size2,
	            D_d, size1,
	      1.0,  u_d, size2);

  gpuErrchk(hipMemcpy(u, u_d, size3*sizeof(double), hipMemcpyDeviceToHost))
  
  gpuErrchk(hipFree(ur_d)) gpuErrchk(hipFree(us_d)) gpuErrchk(hipFree(ut_d)) gpuErrchk(hipFree(u_d))
  gpuErrchk(hipFree(D_d)) gpuErrchk(hipFree(Dt_d))
}

static __global__ void transform_k(double* ur,
                                   double* us,
				   double* ut,
				   double* trans,
				   int n){

  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  const int nthreads = blockDim.x * gridDim.x;
  double wr, ws, wt;
  int i;

  for (i = idx; i < n; i += nthreads){
    wr = trans[6*i+0]*ur[i] + trans[6*i+1] * us[i] + trans[6*i+2]*ut[i];
    ws = trans[6*i+1]*ur[i] + trans[6*i+3] * us[i] + trans[6*i+4]*ut[i];
    wt = trans[6*i+2]*ur[i] + trans[6*i+4] * us[i] + trans[6*i+5]*ut[i];
    ur[i] = wr;
    us[i] = ws;
    ut[i] = wt;
  }
}

static __global__ void set_addr(double** batch_u,
                                double** batch_us,
				double** batch_Dt,
				double** batch_D,
				double* u,
				double* us,
				double* Dt,
				double* D,
				int size,
				int num){
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  const int nthreads = blockDim.x * gridDim.x;
  int i;
  for (i = idx; i < num; i+=nthreads){
    batch_u[i] = u + i*size;
    batch_us[i] = us + i*size;
    batch_Dt[i] = Dt;
    batch_D[i] = D;
  }
}

extern "C" void local_grad3_comb_cuda_(double *w,
                                      double* u,
                                      double* D,
				      double* Dt,
				      double* g,
				      int* n){
  int size1 = *n + 1;
  int size2 = size1 * size1;
  int size3 = size1 * size1 * size1;
  int i;

  double *u_d, *D_d, *Dt_d, *g_d;
  double *ur_d, *us_d, *ut_d; 

  gpuErrchk(hipMalloc(&ur_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&us_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&ut_d, size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&u_d , size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&g_d , 6 * size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&D_d , size2 * sizeof(double)))
  gpuErrchk(hipMalloc(&Dt_d, size2 * sizeof(double)))

  gpuErrchk(hipMemcpy(u_d , u , size3*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(D_d , D , size2*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(Dt_d, Dt, size2*sizeof(double), hipMemcpyHostToDevice))
  gpuErrchk(hipMemcpy(g_d , g , 6*size3*sizeof(double), hipMemcpyHostToDevice))

  // dgemm('N','N',n1,n3,n2,1.0,a,n1,b,n2,0.0,c,n1)
  magma_dgemm('N', 'N', size1, size2, size1, 
              1.0,  D_d, size1, 
	            u_d, size1,
              0.0, ur_d, size1);
  for (i = 0; i < size1; i++){
    magma_dgemm('N', 'N', size1, size1, size1,
                1.0,  u_d + i*size2, size1,
		     Dt_d          , size1,
		0.0, us_d + i*size2, size1);
  }
  magma_dgemm('N', 'N', size2, size1, size1,
              1.0, u_d , size2,
	           Dt_d, size1,
	      0.0, ut_d, size2);

  transform_k<<<128, 256>>>(ur_d, us_d, ut_d, g_d, size3);

  magma_dgemm('N', 'N', size1, size2, size1, 
              1.0, Dt_d, size1, 
	           ur_d, size1,
              0.0,  u_d, size1);
  for (i = 0; i < size1; i++){
    magma_dgemm('N', 'N', size1, size1, size1,
                1.0, us_d + i*size2, size1,
		      D_d          , size1,
		1.0,  u_d + i*size2, size1);
  }
  magma_dgemm('N', 'N', size2, size1, size1,
              1.0, ut_d, size2,
	            D_d, size1,
	      1.0,  u_d, size2);

  gpuErrchk(hipMemcpy(w, u_d, size3*sizeof(double), hipMemcpyDeviceToHost))
  
  gpuErrchk(hipFree(ur_d)) gpuErrchk(hipFree(us_d)) gpuErrchk(hipFree(ut_d)) gpuErrchk(hipFree(u_d))
  gpuErrchk(hipFree(D_d)) gpuErrchk(hipFree(Dt_d)) gpuErrchk(hipFree(g_d))
}

#define NUM_BLOCK_MAX 1
#define NUM_STREAM_MAX 4
#define USE_BATCH
extern hipStream_t* streams;
extern hipblasHandle_t cublas_ctx;
extern "C" void ax_e_cuda_(double *w,
                                      double* u,
                                      double* D,
				      double* Dt,
				      double* g,
				      int* n,
				      int* m){
  int size1 = *n + 1;
  int size2 = size1 * size1;
  int size3 = size1 * size1 * size1;
  int num = *m;
  int i, j, jp;

  int num_block_l;
 
  const double zero = 0.0, one = 1.0;

  double *u_d, *D_d, *Dt_d, *g_d;
  double *ur_d, *us_d, *ut_d; 
  double *u_l, *ur_l, *us_l, *ut_l, *g_l;
  double *u_h, *w_h, *g_h;

  double **batch_Dt_d; double **batch_D_d;
  double **batch_u_d; double **batch_us_d; 
  double **batch_u_l; double **batch_us_l; 

  // First, copy over D, Dt
  gpuErrchk(hipMalloc(&D_d , size2 * sizeof(double)))
  gpuErrchk(hipMalloc(&Dt_d, size2 * sizeof(double)))
  gpuErrchk(hipMemcpy(D_d , D , size2*sizeof(double), hipMemcpyHostToDevice))
#if 1
  gpuErrchk(hipMemcpy(Dt_d, Dt, size2*sizeof(double), hipMemcpyHostToDevice))
#else
  gpuErrchk(hipMemcpy(Dt_d, D_d, size2*sizeof(double), hipMemcpyDeviceToDevice)) 
  magmablas_dtranspose_inplace(size1, Dt_d, size1);
#endif
  // Allocate space for other stuff
  gpuErrchk(hipMalloc(&ur_d, NUM_BLOCK_MAX * NUM_STREAM_MAX * size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&us_d, NUM_BLOCK_MAX * NUM_STREAM_MAX * size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&ut_d, NUM_BLOCK_MAX * NUM_STREAM_MAX * size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&u_d , NUM_BLOCK_MAX * NUM_STREAM_MAX * size3 * sizeof(double)))
  gpuErrchk(hipMalloc(&g_d , NUM_BLOCK_MAX * NUM_STREAM_MAX * 6 * size3 * sizeof(double)))

  gpuErrchk(hipMalloc(&batch_u_d , NUM_BLOCK_MAX * NUM_STREAM_MAX * size1 * sizeof(double*)))
  gpuErrchk(hipMalloc(&batch_us_d , NUM_BLOCK_MAX * NUM_STREAM_MAX * size1 * sizeof(double*)))
  gpuErrchk(hipMalloc(&batch_Dt_d , NUM_BLOCK_MAX * size1 * sizeof(double*)))
  gpuErrchk(hipMalloc(&batch_D_d , NUM_BLOCK_MAX * size1 * sizeof(double*)))

  // Pin the input/output buffers in-place
  u_h = u; w_h = w; g_h = g;
  gpuErrchk(hipHostRegister(u_h, num*size3*sizeof(double), 0));
  gpuErrchk(hipHostRegister(w_h, num*size3*sizeof(double), 0));
  gpuErrchk(hipHostRegister(g_h, num*6*size3*sizeof(double), 0));

  for (j = 0; j < num; j+=NUM_BLOCK_MAX){ 
    jp = (j/NUM_BLOCK_MAX) % NUM_STREAM_MAX;
    num_block_l = MIN(NUM_BLOCK_MAX, num-j*NUM_BLOCK_MAX);
    u_l = u_d + jp*size3; ur_l = ur_d + jp*size3; us_l = us_d + jp*size3; ut_l = ut_d + jp*size3;
    g_l = g_d + jp*6*size3;
    batch_u_l = batch_u_d + jp*size1; batch_us_l = batch_us_d + jp*size1; 
    gpuErrchk(hipMemcpyAsync(u_l, u_h+  j*size3,   size3*sizeof(double)*num_block_l, hipMemcpyHostToDevice, streams[jp]))
    gpuErrchk(hipMemcpyAsync(g_l, g_h+6*j*size3, 6*size3*sizeof(double)*num_block_l, hipMemcpyHostToDevice, streams[jp]))

#ifdef USE_BATCH
    set_addr<<<1,32, 0, streams[jp]>>>(batch_u_l, batch_us_l, batch_Dt_d, batch_D_d,
                                       u_l, us_l, Dt_d, D_d, 
				       size2, size1*num_block_l);
#endif
    // dgemm('N','N',n1,n3,n2,1.0,a,n1,b,n2,0.0,c,n1)
    magmablasSetKernelStream(streams[jp]);
    hipblasSetStream(cublas_ctx, streams[jp]);
    magma_dgemm('T', 'N', size1, size2*num_block_l, size1, 
                1.0,  Dt_d, size1, 
                      u_l, size1,
                0.0, ur_l, size1);
#ifdef USE_BATCH
    hipblasDgemmBatched(cublas_ctx,
                       HIPBLAS_OP_N, HIPBLAS_OP_N, size1, size1, size1,
		       &one, (const double **) batch_u_l, size1,
		             (const double **) batch_Dt_d, size1,
		       &zero, batch_us_l, size1,
		           size1*num_block_l);
#else
    for (i = 0; i < size1*num_block_l; i++){
      magma_dgemm('N', 'N', size1, size1, size1,
                  1.0,  u_l + i*size2, size1,
                       Dt_d          , size1,
                  0.0, us_l + i*size2, size1);
    }
#endif
    for (i = 0;i < num_block_l; i++){
    magma_dgemm('N', 'N', size2, size1, size1,
                1.0, u_l +i*size3, size2,
                     Dt_d, size1,
	        0.0, ut_l +i*size3, size2);
    }

    transform_k<<<128, 256, 0, streams[jp]>>>(ur_l, us_l, ut_l, g_l, size3*num_block_l);

    magma_dgemm('T', 'N', size1, size2*num_block_l, size1, 
                1.0, D_d, size1, 
                     ur_l, size1,
                0.0,  u_l, size1);
#ifdef USE_BATCH
    hipblasDgemmBatched(cublas_ctx,
                       HIPBLAS_OP_N, HIPBLAS_OP_N, size1, size1, size1,
		       &one, (const double **) batch_us_l, size1,
		             (const double **) batch_D_d, size1,
		       &one, batch_u_l, size1,
		             size1*num_block_l);
#else
    for (i = 0; i < size1*num_block_l; i++){
      magma_dgemm('N', 'N', size1, size1, size1,
                  1.0, us_l + i*size2, size1,
                        D_d          , size1,
	          1.0,  u_l + i*size2, size1);
    }
#endif
    for (i = 0; i < num_block_l; i++){
    magma_dgemm('N', 'N', size2, size1, size1,
                1.0, ut_l + i*size3, size2,
                      D_d, size1,
	        1.0,  u_l + i*size3, size2);
    }
    gpuErrchk(hipMemcpyAsync(w_h+j*size3, u_l, size3*sizeof(double)*num_block_l, hipMemcpyDeviceToHost, streams[jp]))
  }
  hipDeviceSynchronize(); 

  // Unpin the memcpy buffers
  gpuErrchk(hipHostUnregister(u_h));
  gpuErrchk(hipHostUnregister(w_h));
  gpuErrchk(hipHostUnregister(g_h));

  gpuErrchk(hipFree(ur_d)) gpuErrchk(hipFree(us_d)) gpuErrchk(hipFree(ut_d)) gpuErrchk(hipFree(u_d))
  gpuErrchk(hipFree(D_d)) gpuErrchk(hipFree(Dt_d)) gpuErrchk(hipFree(g_d))
  gpuErrchk(hipFree(batch_u_d)) gpuErrchk(hipFree(batch_us_d)) gpuErrchk(hipFree(batch_Dt_d)) gpuErrchk(hipFree(batch_D_d))
}

