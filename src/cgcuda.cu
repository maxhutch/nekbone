#include "hip/hip_runtime.h"

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "cuda_util.h"
#include "hipblas.h"
#include "ld_functions.h"
#include "sm_utils.inl"
#include "thrust/device_ptr.h"
#include "thrust/reduce.h"
#include "mpi.h"

#include "cgcuda.h"

hipblasHandle_t cublas_handle;

//#define AOS

struct comm_data {
  uint n;      /* number of messages */
  uint *p;     /* message source/dest proc */
  uint *size;  /* size of message */
  uint total;  /* sum of message sizes */
};

struct mpi_comm_wrapper 
{
 MPI_Comm mpi_comm;
 uint id;
 uint np;
};

struct gpu_map 
{
  uint size_from;
  uint size_to;
  uint* d_offsets;
  uint* d_indices_from;
  uint* d_indices_from_COO;
  uint* d_indices_to;
};

struct gpu_domain
{
  double *d_x; 
  double *d_f; 
  double *d_g; // metric terms
  double *d_c;
  double *d_r;
  double *d_w;
  double *d_p; 
  double *d_z;
  double *d_dxm1; // differentiation matrix D
  double *d_dxtm1; // D^T
  double *d_temp; // temporary array

  double *reduced_value;
  hipEvent_t reduced_value_event;

  int nx1; 
  int ny1; 
  int nz1; 
  int nelt; 
  int ldim; 
  int nxyz;
  int nid;
  int niter;

  struct gpu_map local_map[2];
  uint* d_flagged_primaries;
  int flagged_primaries_size;

  // MPI communication 
  struct comm_data comm_struct[2];
  struct gpu_map comm_map[2];

  // Mpi buffer
  uint buffer_size;
  double* h_buffer;
  double* d_buffer;

  // Mpi requests
  MPI_Request* req; 
  mpi_comm_wrapper comm;
};

static gpu_domain gpu_dom;

void init_comm_struct(comm_data* comm_struct, uint n, const uint* p, const uint* size, uint total)
{
  comm_struct->p = (uint*) malloc(2*n*sizeof(uint));
  comm_struct->size = comm_struct->p + n;

  for (int i=0;i<n;i++)
  {
    (comm_struct->p)[i] = p[i];
    (comm_struct->size)[i] = size[i];
  }

  comm_struct->n = n;
  comm_struct->total = total;
}

void fill_gpu_maps(gpu_map* cuda_map, const uint* map)
{
  cudaCheckError();
  const uint* orig = map;

  // First compute the size of the map_indices_to and map_indices_from arrays
  // TODO: Can probably get that from "nz" array
  uint size_from_tmp = 0;
  uint size_to_tmp = 0;
  while( *map++ != -(uint)1 )                                              
  {
    *map++;  
    size_from_tmp++;
    do { size_to_tmp++; } while( (*map++) != -(uint)1 ); 
  } 
#ifdef DEBUG
  printf("size_from = %d, size_to=%d\n",size_from_tmp,size_to_tmp);
#endif

  // Fill host arrays first
  uint* h_map_offsets = (uint*) malloc((size_from_tmp+1)*sizeof(uint));
  uint* h_map_indices_from = (uint*) malloc(size_from_tmp*sizeof(uint));
  uint* h_map_indices_to = (uint*) malloc(size_to_tmp*sizeof(uint));
  uint* h_map_indices_from_COO = (uint*) malloc(size_to_tmp*sizeof(uint));

  uint i,j;
  uint count_from=0;
  uint count_to=0;
  h_map_offsets[0] = 0;

  cudaCheckError();
  map = orig;
  while((i=*map++)!=-(uint)1) 
  { 
    uint row_length = 0;
    h_map_indices_from[count_from++] = i;
    j=*map++; 
    do 
    { 
      h_map_indices_to[count_to] = j;
      h_map_indices_from_COO[count_to++] = i;
      row_length++;
    }
    while((j=*map++)!=-(uint)1);
  
    h_map_offsets[count_from] = count_to; 
  } 

//   printf("size_from=%d\n",size_from_tmp);
//   printf("size_to=%d\n",size_to_tmp);
//   for (int i = 0; i < size_from_tmp; i++) {
//     printf("%d: ", h_map_indices_from[i]);
//     for (int j = h_map_offsets[i]; j < h_map_offsets[i+1]; j++)
//       printf("%d ", h_map_indices_to[j]);
//     printf("\n");
//   }
//   printf("\n");
//   for (int i = 0; i < size_to_tmp; i++)
//     printf("%d ", h_map_indices_to[i]);
//   printf("\n");

  cudaCheckError();
  hipMalloc((void **) &(cuda_map->d_offsets), (size_from_tmp+1)*sizeof(uint));
  hipMalloc((void **) &(cuda_map->d_indices_from), size_from_tmp*sizeof(uint));
  hipMalloc((void **) &(cuda_map->d_indices_to), size_to_tmp*sizeof(uint));
  hipMalloc((void **) &(cuda_map->d_indices_from_COO), size_to_tmp*sizeof(uint));

  hipMemcpy(cuda_map->d_offsets, h_map_offsets, (size_from_tmp+1)*sizeof(uint), hipMemcpyHostToDevice);
  hipMemcpy(cuda_map->d_indices_from, h_map_indices_from, size_from_tmp*sizeof(uint), hipMemcpyHostToDevice);
  hipMemcpy(cuda_map->d_indices_to, h_map_indices_to, size_to_tmp*sizeof(uint), hipMemcpyHostToDevice);
  hipMemcpy(cuda_map->d_indices_from_COO, h_map_indices_from_COO, size_to_tmp*sizeof(uint), hipMemcpyHostToDevice);

  cuda_map->size_from = size_from_tmp;
  cuda_map->size_to = size_to_tmp;
  cudaCheckError();
}

void fill_flagged_primaries_map(uint** d_flagged_primaries, const uint* flagged_primaries)
{
//  hipMalloc((void **) d_flagged_primaries, (size_from_tmp+1)*sizeof(uint));
}

template <typename T>
__global__
void local_init_kernel(T* __restrict__ out,  const uint* __restrict__ flagged_primaries, int size  )
{
  for (int tid = blockDim.x*blockIdx.x + threadIdx.x; tid < size; tid+= gridDim.x * blockDim.x) 
  {
    out[flagged_primaries[tid]] = 0.;
  }
}


template <typename T>
__global__
void local_gather_kernel(T* __restrict__ out, const T* __restrict__ in, 
                         const uint* __restrict__ offsets, 
                         const uint* __restrict__ map_indices_from, 
                         const uint* __restrict__ map_indices_to, int size  )
{
  for (int tid = blockDim.x*blockIdx.x + threadIdx.x; tid < size; tid+= gridDim.x * blockDim.x) 
  {
    uint store_loc = map_indices_from[tid];
    T t = out[store_loc]; 
    for (int i=offsets[tid];i<offsets[tid+1];i++)
    {
      t += in[map_indices_to[i]];
    }
    out[store_loc] = t;
  }
}

template <typename T>
__global__
void local_scatter_kernel(T* __restrict__ out, const T* __restrict__ in, 
                          const uint* __restrict__ offsets, 
                          const uint* __restrict__ map_indices_from, 
                          const uint* __restrict__ map_indices_to, int size  )
{
  for (int tid = blockDim.x*blockIdx.x + threadIdx.x; tid < size; tid+= gridDim.x * blockDim.x) 
  {
    T t = in[map_indices_from[tid]]; 
    for (int i=offsets[tid];i<offsets[tid+1];i++)
    {
      out[map_indices_to[i]] = t;  
    }
  }
}


template <typename T>
__global__
void local_scatter_kernel_COO(T* __restrict__ out, const T* __restrict__ in,  const uint* __restrict__ map_indices_from_COO, const uint* __restrict__ map_indices_to, int size  )
{
  for (int tid = blockDim.x*blockIdx.x + threadIdx.x; tid < size; tid+= gridDim.x * blockDim.x) 
  {
    T t = in[map_indices_from_COO[tid]]; 
    out[map_indices_to[tid]] = t;  
  }
}

void local_init_cuda(double* out, uint* flagged_primaries, int flagged_primaries_size)
//                       const uint *map_offsets, const uint* map_indices_from, const uint* map_indices_from_COO, const uint* map_indices_to, int size_from, int size_to)
{

  const int cta_size= 128;
  const int grid_size = min(4096,(flagged_primaries_size+cta_size-1)/cta_size);
  cudaCheckError();
  local_init_kernel<<<grid_size,cta_size>>>(out,flagged_primaries,flagged_primaries_size);
  cudaCheckError();
}




void local_gather_cuda(double* out, const double* in,  gpu_map* map)
//                       const uint *map_offsets, const uint* map_indices_from, const uint* map_indices_from_COO, const uint* map_indices_to, int size_from, int size_to)
{

  cudaCheckError();

  const int cta_size= 128;
  const int grid_size = min(4096,(map->size_from+cta_size-1)/cta_size);
  cudaCheckError();
  local_gather_kernel<<<grid_size,cta_size>>>(out,in,map->d_offsets,
                                              map->d_indices_from,map->d_indices_to,map->size_from);
  cudaCheckError();
}



//void local_scatter_cuda(double* out, const double* in,
//                       const uint *map_offsets, const uint* map_indices_from, const uint* map_indices_from_COO, const uint* map_indices_to, int size_from, int size_to)
//
void local_scatter_cuda(double* out, double* in, gpu_map* map)
//  pdouble* out, const double* in,
//                       const uint *map_offsets, const uint* map_indices_from, const uint* map_indices_from_COO, const uint* map_indices_to, int size_from, int size_to)
{

  const int cta_size= 128;
  const int grid_size = min(4096,(map->size_from+cta_size-1)/cta_size);
  //const int grid_size = min(1,(map->size_from+cta_size-1)/cta_size);
  local_scatter_kernel<<<grid_size,cta_size>>>(out,in,map->d_offsets,
                                               map->d_indices_from,map->d_indices_to,map->size_from );
  cudaCheckError();

  //const int cta_size= 128;
  //const int grid_size = min(4096,(map->size_to+cta_size-1)/cta_size);
  //local_scatter_kernel_COO<<<grid_size,cta_size>>>(out,in,map->d_indices_from_COO,map->d_indices_to,map->size_to);
  //cudaCheckError();

}

__global__
void mask_kernel(double* w, int size)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  if (tid < size)
  {
    w[tid] = 0.;
  }
}

#if 0
  /*
   * Copy data to the GPU
  */
  void copyTo(int nelt, int nxyz, double *u) {

    cudaCheckError();
    hipMemcpy(d_u, u, nelt*nxyz*sizeof(double), hipMemcpyHostToDevice);
    cudaCheckError();

    return; 
  }


  /*
   * Copy data from the GPU
  */
template<typename T>
  void copyFrom(int n, T *vec, T *d_vec) {

    cudaCheckError();
    hipMemcpy(vec, d_vec, n*sizeof(T), hipMemcpyDeviceToHost);
    cudaCheckError();

    return;
  }
#endif


  /*
   * Stacked Matrix multiply
  */
  __global__ void gsmxm(double *a, int n1, double *b, int n2, double *c, int n3, int nlvl) {
    int id= blockDim.x*blockIdx.x+ threadIdx.x;
   
    int aSize= n1*n2; 
    int cSize= n1*n3; 
    int lvl = id/cSize;
    int rank = id % cSize;
    int row = rank % n1;
    int col = rank / n1;

    if (id < cSize*nlvl) {
      c[id] = 0.0;

      int k;
      for (k = 0; k < n2; k++) {
        c[id] += a[lvl*aSize+k*n1+row]*b[col*n2+k];
      }
    }

    return;
  }


  /*
   * Add two vectors
  */
  __global__ void gadd2(double *a, double *b, int n) {
    int id= blockDim.x*blockIdx.x+ threadIdx.x;

    if (id < n) {
      a[id] += b[id];
    }

    return;
  }


  /*
   * Perform geometry scaling
  */
  __global__ void geom(int n, double *ur, double *us, double *ut, double *g) {
    int id= blockDim.x*blockIdx.x+ threadIdx.x;

    if (id < n) {
      double wr = g[id*6+0]*ur[id] + g[id*6+1]*us[id] + g[id*6+2]*ut[id];
      double ws = g[id*6+1]*ur[id] + g[id*6+3]*us[id] + g[id*6+4]*ut[id];
      double wt = g[id*6+2]*ur[id] + g[id*6+4]*us[id] + g[id*6+5]*ut[id];
      ur[id] = wr;
      us[id] = ws;
      ut[id] = wt;
    }

    return;
  }

template<int p, int p_sq, int p_cube, int p_cube_padded, int pts_per_thread, int slab_size, int cta_size, int num_ctas>
__global__
__launch_bounds__(cta_size,num_ctas)
void ax_cuda_kernel_v8_shared_D(const double* __restrict__ u_global, double* __restrict__ w, const double* __restrict__ g, const double* __restrict__ dxm1, const double* __restrict__ dxtm1, int n_cells)
{
  int tid = threadIdx.x;
  __shared__ double temp[cta_size];
  __shared__ double s_dxm1[p_sq];
  __shared__ double s_dxtm1[p_sq];

  for (int cell_id=blockIdx.x; cell_id < n_cells; cell_id += gridDim.x)
  {
    // Load u in shared for the entire cell
    int offset = cell_id*p_cube;

    int tid_mod_p = tid%p;
    int tid_div_p = tid/p;
    int tid_mod_p_sq = tid%p_sq;
    int tid_div_p_sq = tid/p_sq;

    double u[pts_per_thread];
    #pragma unroll
    for (int k=0;k<pts_per_thread;k++)
    {
      int pt_id = k*cta_size + tid;

      u[k] = ld_functions::ld_cg(&u_global[offset + pt_id]);
    }

    // Store dxm1 and dxtm1 in shared memory
    if (tid < p_sq)
    {
      s_dxm1[tid] = ld_functions::ld_cg(&dxm1[tid]);
      s_dxtm1[tid] = ld_functions::ld_cg(&dxtm1[tid]);
    }


    // Initialize wa to 0.
    double wa[pts_per_thread];
    #pragma unroll
    for (int k=0;k<pts_per_thread;k++)
      wa[k] = 0.;

    // Now compute w for one slab at a time
    #pragma unroll
    for (int k=0;k<pts_per_thread;k++)
    {
      int pt_id = k*cta_size + tid;
      //int pt_id_div_p = pt_id/p;
      //int pt_id_mod_p = pt_id%p;
      int pt_id_div_p_sq = pt_id/p_sq;
      //int pt_id_mod_p_sq = pt_id%p_sq;

      double ur, us, ut;

      // Load first slab in shared memory
      __syncthreads();
      temp[tid] = u[k];
      __syncthreads();
      

      //  Now that data is loaded in shared, compute ur
      {
        int s_offset = tid_div_p*p;
        int d_offset  = tid_mod_p;

        ur = 0.;
        #pragma unroll
        for (int i=0;i<p;i++)
          ur += s_dxm1[d_offset + p*i]*temp[s_offset + i];
          //ur += __ldg(&dxm1[d_offset + p*i])*temp[s_offset + i];
      }

      // Compute us
      {
        int plane = tid_div_p_sq;
        int s_offset = plane*p_sq + tid_mod_p;
        int d_offset = p*( (tid-plane*p_sq)/p);

        us = 0.;
        #pragma unroll
        for (int i=0;i<p;i++)
          us += temp[s_offset + p*i]*s_dxtm1[d_offset + i];
         // us += temp[s_offset + p*i]*__ldg(&dxtm1[d_offset + i]);
      }


      // Load all slabs in shared, one by one to compute ut
      ut = 0.;
      #pragma unroll
      for (int k2=0;k2<pts_per_thread;k2++)
      {
        int i_start = k2*slab_size;

        // Load in shared
        __syncthreads();
        temp[tid] = u[k2];
        __syncthreads();

        // Compute ut
        int s_offset = tid_mod_p_sq;
        int d_offset = pt_id_div_p_sq*p;

        #pragma unroll
        for (int icount=0;icount<slab_size;icount++)
        {
          //ut += temp[s_offset + p_sq*icount]*__ldg(&dxtm1[d_offset + i_start]);
          ut += temp[s_offset + p_sq*icount]*s_dxtm1[d_offset + i_start];
          i_start++;
        }
      }

      // Transform
      {


        /*
        int offset = (cell_id*p_cube + pt_id)*6;
        //TODO: Switch to SOA
        double metric[6];
        #pragma unroll
        for (int i=0;i<6;i++)
          metric[i] = g[offset+i];

        */

        // AoS version
#ifdef AOS
        int offset = cell_id*p_cube + pt_id;
        //TODO: Switch to SOA
        double metric[6];
        #pragma unroll
        for (int i=0;i<6;i++)
          metric[i] = g[offset+i*n_cells*p_cube];
#else
        int offset = (cell_id*p_cube + pt_id)*6;
        double metric[6];
        #pragma unroll
        for (int i=0;i<6;i++)
          metric[i] = g[offset+i];
#endif
          //metric[i] = ld_functions::ld_cg(&g[offset+i*n_cells*p_cube]);
          //metric[i] = g[offset+i*n_cells*p_cube];

          //metric[i] = ld_functions::ld_cg(&g[offset+i]);
          //metric[i] = g[offset+i];


        // SOA HACK
        /*
        int offset = cell_id*p_cube + pt_id;

        //TODO: Switch to SOA
        double metric[6];
        #pragma unroll
        for (int i=0;i<6;i++)
        {
          //metric[i] = g[offset];
          metric[i] = ld_functions::ld_cg(&g[offset]);
          offset += n_cells*p_cube;
        }
        */



          //metric[i] = ld_functions::ld_cg(&(g[offset+i]));

        double wr = metric[0]*ur + metric[1]*us + metric[2]*ut;
        double ws = metric[1]*ur + metric[3]*us + metric[4]*ut;
        double wt = metric[2]*ur + metric[4]*us + metric[5]*ut;

        ur = wr;
        us = ws;
        ut = wt;
      }

      // Store ur in shared memory
      __syncthreads();
      temp[tid] = ur;
      __syncthreads();

      // Now that data is loaded in shared, compute wa

      {
        int d_offset  = tid_mod_p;
        int s_offset = tid_div_p*p;

        #pragma unroll
        for (int i=0;i<p;i++)
          wa[k] += s_dxtm1[d_offset+p*i]*temp[s_offset + i];
          //wa[k] += __ldg(&dxtm1[d_offset+p*i])*temp[s_offset + i];
      }

      __syncthreads();
      temp[tid] = us;
      __syncthreads();

      // Compute us
      {

        int plane = tid_div_p_sq;
        int s_offset = plane*p_sq + tid_mod_p;
        int d_offset = p*( (tid-plane*p_sq)/p);

        #pragma unroll
        for (int i=0;i<p;i++)
          wa[k] += temp[s_offset + p*i]*s_dxm1[d_offset + i];
          //wa[k] += temp[s_offset + p*i]*__ldg(&dxm1[d_offset + i]);
      }

      __syncthreads();
      // Store ut in shared memory
      temp[tid] = ut;
      __syncthreads();

      #pragma unroll
      for (int k2=0;k2<pts_per_thread;k2++)
      {
        int i_start = k*slab_size;
        int pt_id_2 = k2*cta_size + tid;
        int plane = pt_id_2/p_sq;

        int s_offset = tid_mod_p_sq;
        int d_offset = plane*p;

        #pragma unroll
        for (int i_count=0; i_count < slab_size; i_count++)
        {
          wa[k2] += temp[s_offset + p_sq*i_count]*s_dxm1[d_offset + i_start];
          //wa[k2] += temp[s_offset + p_sq*i_count]*__ldg(&dxm1[d_offset + i_start]);
          i_start++;
        }
      }
      __syncthreads();

    } // Loop over k

    #pragma unroll
    for (int k=0;k<pts_per_thread;k++)
    {
      int pt_id = k*cta_size + tid;
      w[offset + pt_id] = wa[k];
    }
  } // Loop over blocks

}

//
// Matrix-vector kernel
//
void axcuda_e(double *w, double *u, double *g, double *dxm1, double *dxtm1, 
              int nx1, int ny1, int nz1, int nelt, int ldim) 
{
      if ( nx1 != ny1 || nx1 != nz1)
      {
        printf("non-cubic elements not supported in Cuda version\n");
        exit(1);
      }

      hipDeviceSetCacheConfig(hipFuncCachePreferL1);
      hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

      float time;
      hipEvent_t start,stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);
      hipEventRecord(start, 0);

      //printf("
      if ( nx1 != 12 && nx1 != 9 )
      {
        printf("Current implementation only tested for polynomial orders 9 and 12, exiting");
        exit(1);
      }
      else if ( nx1 == 9 )
      {
        const int grid_size = nelt;
        const int p = 9;
        const int p_sq = 9*9;
        const int p_cube = 9*9*9;
        const int p_cube_padded = p_cube;

        const int cta_size = 243;
        const int pts_per_thread = 3;  // 6*288 = 12*12*12
        const int slab_size = 3;

        const int num_ctas = 4;

        ax_cuda_kernel_v8_shared_D<p,p_sq,p_cube,p_cube_padded,pts_per_thread,slab_size,cta_size,num_ctas>
          <<<grid_size,cta_size>>>(u, w, g, dxm1, dxtm1, nelt);
      }
      else if ( nx1 == 12 )
      {
        // 12x12x12 case
        const int p = 12;
        const int p_sq = 12*12;
        const int p_cube = 12*12*12;
        const int p_cube_padded = p_cube;

        // We could play with this
        const int grid_size = nelt;

        // BEST CONFIG
        const int cta_size = 576;
        const int pts_per_thread = 3;  // 6*288 = 12*12*12
        const int slab_size = 4;
        const int num_ctas = 2;

        ax_cuda_kernel_v8_shared_D<p,p_sq,p_cube,p_cube_padded,pts_per_thread,slab_size,cta_size,num_ctas>
          <<<grid_size,cta_size>>>(u, w, g, dxm1, dxtm1, nelt);
      }

      cudaCheckError();
      hipEventRecord(stop,0);
      hipEventSynchronize(stop);

      hipEventElapsedTime(&time, start, stop);
      return;
}

int exec_mpi_recvs(double *buf, 
                 const mpi_comm_wrapper* comm,
                 const struct comm_data *c, MPI_Request* req)
{
  const uint *p, *pe, *size=c->size;
  int send_size = 0;

  for(p=c->p, pe=p+c->n ; p!=pe ; ++p) 
  {
    size_t len = *(size++);
    MPI_Irecv(buf,len,MPI_DOUBLE,*p,*p,comm->mpi_comm,req++);
    buf += len;
    send_size += len;
  }

  return send_size;
}

int exec_mpi_sends(double *buf, 
                 const mpi_comm_wrapper* comm,
                 const struct comm_data *c, MPI_Request* req)
{
  const uint *p, *pe, *size=c->size;
  int send_size=0;
  for(p=c->p, pe=p+c->n; p!=pe; ++p) 
  {
    size_t len = *(size++);
    MPI_Isend(buf, len, MPI_DOUBLE, *p, comm->id, comm->mpi_comm, req++);

    buf += len;
    send_size+=len;
  }
  return send_size;
}

void exec_mpi_wait(MPI_Request* req, int n)
{
  MPI_Waitall(n,req,MPI_STATUSES_IGNORE);
}


void exec_mpi_reduce_sum(double *val, double* output,
                 const mpi_comm_wrapper* comm)
{
  MPI_Allreduce(val, output, 1, MPI_DOUBLE, MPI_SUM, comm->mpi_comm);
}


void gs_op_cuda(double* w, int dom, int op, int in_transpose)
{
  cudaCheckError();
  bool transpose = (in_transpose!=0);

  const unsigned recv = 0^transpose, send = 1^transpose;

  local_gather_cuda(w,w,&(gpu_dom.local_map[0^transpose]));

  cudaCheckError();
  //local_init_cuda(w,gpu_dom.d_flagged_primaries,gpu_dom.flagged_primaries_size);

  cudaCheckError();
  if (gpu_dom.comm.np > 1)
  {
    // Post mpi receives
    int recv_size = exec_mpi_recvs(gpu_dom.h_buffer, &(gpu_dom.comm), &(gpu_dom.comm_struct[recv]), gpu_dom.req);

    // Fill send buffer
    local_scatter_cuda(gpu_dom.d_buffer+recv_size, w, &(gpu_dom.comm_map[send]));

    // Copy buffer from device to host
    hipMemcpy(gpu_dom.h_buffer+recv_size, gpu_dom.d_buffer+recv_size, 
               gpu_dom.comm_map[send].size_to*sizeof(double), hipMemcpyDeviceToHost);

    // Send host buffer
    exec_mpi_sends( gpu_dom.h_buffer+recv_size, &(gpu_dom.comm), &(gpu_dom.comm_struct[send]), 
                    &(gpu_dom.req[gpu_dom.comm_struct[recv].n ]));

    // Wait for mpi communication to terminate
    exec_mpi_wait(gpu_dom.req, gpu_dom.comm_struct[0].n+gpu_dom.comm_struct[1].n);

    // Copy buffer from host to device
    hipMemcpy(gpu_dom.d_buffer, gpu_dom.h_buffer, recv_size*sizeof(double), hipMemcpyHostToDevice);

    // Gather from buffer
    local_gather_cuda(w, gpu_dom.d_buffer, &(gpu_dom.comm_map[recv]) );
  }

  local_scatter_cuda(w, w, &(gpu_dom.local_map[1^transpose]) );
}

void add2s2_cuda(double *a, double *b, double c1, int n)
{
  hipblasDaxpy(cublas_handle, n, &c1, b, 1, a, 1);
  cudaCheckError();
}


__global__ void add2s1_kernel(double *a, double *b, double c1, int n)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if (threadIdx.x < n) {
    a[tid] = c1*a[tid] + b[tid];
  }
}

void add2s1_cuda(double *a, double *b, double c1, int n)
{

 // TODO: should probably merge into single kernel

//   double one = 1.;
//   hipblasDscal(cublas_handle, n, &c1, a, 1); 
//   cudaCheckError();
//   hipblasDaxpy(cublas_handle, n, &one, b, 1, a, 1);
//   cudaCheckError();

  int block = 256;
  int grid = (n + block - 1) / block;
  add2s1_kernel<<<grid, block>>>(a, b, c1, n);
  cudaCheckError();
}


void mask_cuda(double *w, int nid)
{
  if (nid == 0)
  {
    mask_kernel<<<1,1>>>(w,1);
  }
}

void rzero_cuda(double *a, int n)
{
  double zero = 0.;
  cudaCheckError();
  hipblasDscal(cublas_handle, n, &zero, a, 1);
  cudaCheckError();
}

void copy_cuda(double *a, double* b, int n)
{
  cudaCheckError();
  hipblasDcopy(cublas_handle, n, b, 1, a, 1); 
  cudaCheckError();
}


__global__ 
void glsc3_cuda_kernel(double* a, double* b, double* mult, double* result, int n)
{
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  while (tid < n)
  {
    result[tid] = a[tid]*b[tid]*mult[tid];
    tid += gridDim.x*blockDim.x;
  }
}

double glsc3_cuda(double *a, double* b, double* mult,  int n)
{
  cudaCheckError();

  double glsc3;
  // TODO: This could be made faster by having a single kernel followed by reduction across block, 
  // instead of writing to global memory
  int cta_size = 128;
  int grid_size=min(4096,( (n+cta_size-1)/cta_size));

  glsc3_cuda_kernel<<<grid_size,cta_size>>>(a,b,mult,gpu_dom.d_temp,n);

  thrust::device_ptr<double> beg(gpu_dom.d_temp);
  glsc3 = thrust::reduce(beg,beg+n);
  
 // printf("before reduce %e\n",glsc3);

  exec_mpi_reduce_sum(&glsc3,&glsc3,&(gpu_dom.comm));

  //printf("after educe %e\n",glsc3);

  cudaCheckError();
  return glsc3;
}

void solveM_cuda(double *z, double* r, int n)
{
  copy_cuda(z,r,n);
}

void axcuda(double* w, double* u, double *g, double *dxm1, double* dxtm1, 
            int nx1, int ny1, int nz1, int nelt, int ldim, int nid, 
            double *flop_a)
{

  axcuda_e(w,u,g,dxm1,dxtm1,nx1,ny1,nz1,nelt,ldim);

  // TODO: Currently, parameters dom and op are ignored
  gs_op_cuda(w,1,1,0); 

  int n = nx1*ny1*nz1*nelt;
  add2s2_cuda(w,u,.1,n);
  mask_cuda(w,nid);

  int nxyz = nx1*ny1*nz1;
  *flop_a += (19*nxyz+12*nx1*nxyz)*nelt;
}


extern "C"
{
  void gs_setup_cuda(const uint* map_local_0, const uint* map_local_1, const uint* flagged_primaries)
  {
    // Initialize data required for gather-scatter operation
    fill_gpu_maps( &(gpu_dom.local_map[0]), map_local_0);
    fill_gpu_maps( &(gpu_dom.local_map[1]), map_local_1);

    fill_flagged_primaries_map(&(gpu_dom.d_flagged_primaries),flagged_primaries);
  }

  void gs_comm_setup_cuda(const uint comm_0_n, const uint* comm_0_p, const uint* comm_0_size, const uint comm_0_total,
                          const uint comm_1_n, const uint* comm_1_p, const uint* comm_1_size, const uint comm_1_total,
                          const uint* map_comm_0, const uint* map_comm_1,
                          uint buffer_size,
                          const MPI_Comm* mpi_comm,
                          int comm_id,
                          int comm_np)
  {
    // Duplicate the MPI communicator
    MPI_Comm_dup( (*mpi_comm), &(gpu_dom.comm.mpi_comm) );
    gpu_dom.comm.id = comm_id;
    gpu_dom.comm.np = comm_np;

    if (gpu_dom.comm.np > 1)
    {
      // Initialize the communication structure
      init_comm_struct( &(gpu_dom.comm_struct[0]), comm_0_n, comm_0_p, comm_0_size, comm_0_total);
      init_comm_struct( &(gpu_dom.comm_struct[1]), comm_1_n, comm_1_p, comm_1_size, comm_1_total);
      cudaCheckError();

      // Create the MPI gather-scatter maps
      fill_gpu_maps( &(gpu_dom.comm_map[0]), map_comm_0);
      fill_gpu_maps( &(gpu_dom.comm_map[1]), map_comm_1);
      cudaCheckError();

      // Allocate the send and receive buffers on host and device
      gpu_dom.buffer_size = buffer_size;
      gpu_dom.h_buffer= (double*) malloc( (buffer_size)*sizeof(double));
      hipMalloc((double **) &(gpu_dom.d_buffer), buffer_size*sizeof(double));
      cudaCheckError();

      // Create the array of MPI requests
      gpu_dom.req = (MPI_Request*) malloc( (comm_0_n+comm_1_n)*sizeof(MPI_Request));
      cudaCheckError();
    }
  }

  void cg_cuda_set_device_(int* nid)
  {
    int gpu_count;
    hipGetDeviceCount(&gpu_count);
    int device_id = (*nid)%gpu_count;
    hipSetDevice(device_id);
    printf("rank %d selecting gpu %d\n",*nid,device_id);
  }

  void cg_cuda_init_(double* x, double* f, double* g, double* c, double* r, double* w, double* p, double* z, int* nx1, int* ny1, int* nz1, int* nelt, int* ldim, double* dxm1, double* dxtm1, int* niter, double* flop_cg, const int *gsh_handle, int* nid)
  {
    // Initialize gpu_dom structure
    // Note: the gsh structure on the GPU is already initialized, since gs_cuda_setup is called in the proxy_setupds function in driver.f

    gpu_dom.nid = (*nid);
    gpu_dom.nx1 = (*nx1);
    gpu_dom.ny1 = (*ny1);
    gpu_dom.nz1 = (*nz1);
    gpu_dom.ldim= (*ldim);
    gpu_dom.nelt = (*nelt);
    gpu_dom.niter = (*niter);

    int nxyz = (*nx1)*(*ny1)*(*nz1);
    
    // Initializing the Cublas library 
    if (cublas_handle==NULL)
      hipblasCreate(&cublas_handle);

    hipEventCreateWithFlags(&(gpu_dom.reduced_value_event),hipEventDisableTiming);
    cudaCheckError();

    // malloc GPU memory for u, w, ur, us, ut, g, dxm1, dxtm1 
    hipMalloc((void **)&gpu_dom.d_w, gpu_dom.nelt*nxyz*sizeof(double));
    hipMalloc((void **)&gpu_dom.d_f, gpu_dom.nelt*nxyz*sizeof(double));
    hipMalloc((void **)&gpu_dom.d_c, gpu_dom.nelt*nxyz*sizeof(double));
    hipMalloc((void **)&gpu_dom.d_r, gpu_dom.nelt*nxyz*sizeof(double));
    hipMalloc((void **)&gpu_dom.d_p, gpu_dom.nelt*nxyz*sizeof(double));
    hipMalloc((void **)&gpu_dom.d_z, gpu_dom.nelt*nxyz*sizeof(double));
    hipMalloc((void **)&gpu_dom.d_x, gpu_dom.nelt*nxyz*sizeof(double));
    hipMalloc((void **)&gpu_dom.d_temp, gpu_dom.nelt*nxyz*sizeof(double));
    hipMalloc((void **)&gpu_dom.d_g, gpu_dom.nelt*nxyz*2*gpu_dom.ldim*sizeof(double));
    hipMalloc((void **)&gpu_dom.d_dxm1, gpu_dom.nx1*gpu_dom.nx1*sizeof(double));
    hipMalloc((void **)&gpu_dom.d_dxtm1, gpu_dom.nx1*gpu_dom.nx1*sizeof(double));

    hipHostMalloc(&gpu_dom.reduced_value,sizeof(double));

    cudaCheckError();

    // copy data to the GPU

    hipMemcpy(gpu_dom.d_w, w, gpu_dom.nelt*nxyz*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_dom.d_f, f, gpu_dom.nelt*nxyz*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_dom.d_c, c, gpu_dom.nelt*nxyz*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_dom.d_r, r, gpu_dom.nelt*nxyz*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_dom.d_p, p, gpu_dom.nelt*nxyz*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_dom.d_z, z, gpu_dom.nelt*nxyz*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_dom.d_x, x, gpu_dom.nelt*nxyz*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_dom.d_p, p, gpu_dom.nelt*nxyz*sizeof(double), hipMemcpyHostToDevice);


    // Switch to AoS for d_g
#ifdef AOS
    double* g_tmp = (double*) malloc(gpu_dom.nelt*nxyz*2*gpu_dom.ldim*sizeof(double));
    int lt = nxyz*gpu_dom.nelt;
    for (int i=0;i<lt;i++)
      for (int j=0;j<6;j++)
        g_tmp[j*lt+i] = g[i*6+j];
    hipMemcpy(gpu_dom.d_g, g_tmp, gpu_dom.nelt*nxyz*2*gpu_dom.ldim*sizeof(double), hipMemcpyHostToDevice);
#else
    hipMemcpy(gpu_dom.d_g, g, gpu_dom.nelt*nxyz*2*gpu_dom.ldim*sizeof(double), hipMemcpyHostToDevice);
#endif

    hipMemcpy(gpu_dom.d_dxm1, dxm1, gpu_dom.nx1*gpu_dom.nx1*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_dom.d_dxtm1, dxtm1, gpu_dom.nx1*gpu_dom.nx1*sizeof(double), hipMemcpyHostToDevice);

    cudaCheckError();

    return;
  }


  void cg_cuda_(double* r, int* copy_to_cpu, double *flop_cg, double *flop_a)
  {
    int n = gpu_dom.nx1 * gpu_dom.ny1 * gpu_dom.nz1 * gpu_dom.nelt;
    double pap = 0.0;

    // set machine tolerances
    double one = 1.;
    double eps = 1.e-20;

    if (one+eps == one) 
      eps = 1.e-14;
    if (one+eps == one) 
      eps = 1.e-7;

    double rtz1 = 1.0;

    cudaCheckError();
    rzero_cuda(gpu_dom.d_x, n);
    copy_cuda(gpu_dom.d_r, gpu_dom.d_f, n);

    // Zero out Dirichlet conditions
    mask_cuda(gpu_dom.d_r, gpu_dom.nid);

    double rnorm = sqrt(glsc3_cuda(gpu_dom.d_r, gpu_dom.d_c, gpu_dom.d_r, n));
      
    int iter = 0;
    if (gpu_dom.nid==0) 
      printf("cg:%4d %11.4e\n",iter,rnorm);

    int miter = gpu_dom.niter;
    double alpha, beta;

    for (iter=1; iter <= miter; iter++)
    {
       solveM_cuda(gpu_dom.d_z, gpu_dom.d_r, n);

       double rtz2=rtz1;
       rtz1 = glsc3_cuda(gpu_dom.d_r, gpu_dom.d_c, gpu_dom.d_z, n);

       beta = rtz1/rtz2;
       if (iter==1) 
         beta=0.0;

#ifdef DEBUG
       printf("rtz1 = %12.8g\n",rtz1);
#endif
       add2s1_cuda(gpu_dom.d_p, gpu_dom.d_z, beta, n);

       axcuda(gpu_dom.d_w, gpu_dom.d_p, gpu_dom.d_g, gpu_dom.d_dxm1, gpu_dom.d_dxtm1, 
              gpu_dom.nx1, gpu_dom.ny1, gpu_dom.nz1, gpu_dom.nelt, gpu_dom.ldim, gpu_dom.nid,
              flop_a);

       pap = glsc3_cuda(gpu_dom.d_w, gpu_dom.d_c, gpu_dom.d_p, n);
#ifdef DEBUG
       printf("pap = %12.8g\n",pap);
#endif
       alpha=rtz1/pap;
       double alphm = -alpha;

       add2s2_cuda(gpu_dom.d_x, gpu_dom.d_p, alpha, n);
       add2s2_cuda(gpu_dom.d_r, gpu_dom.d_w, alphm, n);

       double rtr = glsc3_cuda(gpu_dom.d_r, gpu_dom.d_c, gpu_dom.d_r, n);

       rnorm = sqrt(rtr);
    }

    *flop_cg += miter*15.0*n + 3.0*n;

    if (gpu_dom.nid==0) 
      printf("cg:%4d %11.4e %11.4e %11.4e %11.4e\n",iter,rnorm,alpha,beta,pap);

    if ((*copy_to_cpu)==1)
      hipMemcpy(r, gpu_dom.d_r, n*sizeof(double), hipMemcpyDeviceToHost);

  }

  void cg_cuda_free_() 
  {
    /* free GPU memory for u, w, ur, us, ut, g, dxm1, dxtm1  */

    cudaCheckError();

    hipFree(gpu_dom.local_map[0].d_offsets);
    hipFree(gpu_dom.local_map[0].d_indices_from);
    hipFree(gpu_dom.local_map[0].d_indices_from_COO);
    hipFree(gpu_dom.local_map[0].d_indices_to);

    hipFree(gpu_dom.local_map[1].d_offsets);
    hipFree(gpu_dom.local_map[1].d_indices_from);
    hipFree(gpu_dom.local_map[1].d_indices_from_COO);
    hipFree(gpu_dom.local_map[1].d_indices_to);

    hipFree(gpu_dom.comm_map[0].d_offsets);
    hipFree(gpu_dom.comm_map[0].d_indices_from);
    hipFree(gpu_dom.comm_map[0].d_indices_from_COO);
    hipFree(gpu_dom.comm_map[0].d_indices_to);

    hipFree(gpu_dom.comm_map[1].d_offsets);
    hipFree(gpu_dom.comm_map[1].d_indices_from);
    hipFree(gpu_dom.comm_map[1].d_indices_from_COO);
    hipFree(gpu_dom.comm_map[1].d_indices_to);

    hipFree(gpu_dom.d_w);
    hipFree(gpu_dom.d_p);
    hipFree(gpu_dom.d_g);
    hipFree(gpu_dom.d_dxm1);
    hipFree(gpu_dom.d_dxtm1);

    cudaCheckError();

    return;
  }

} // extern C





